#include "hip/hip_runtime.h"
//
// Created by wang_shuai on 2020/8/17.
//

#include "customNMS.h"
#include <NvInferRuntimeCommon.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include <array>
#include <hip/hip_fp16.h>

template<typename DataType>
__device__ DataType bbox_size(
        const ONNXBbox<DataType>& bbox
)
{
    return (bbox.xmax - bbox.xmin)*(bbox.ymax - bbox.ymin);
}


template<typename DataType>
__device__ DataType IOU(const ONNXBbox<DataType>&A, ONNXBbox<DataType>&B)
{
    DataType w,h;
    DataType A_area, B_area;
    DataType xmin,ymin,xmax,ymax;
    xmin = max(A.xmin, B.xmin);
    ymin = max(A.ymin, B.ymin);
    xmax = min(A.xmax, B.xmax);
    ymax = min(A.ymax, B.ymax);

    if(xmin > xmax || ymin > ymax) return 0;

    w = xmax - xmin;
    h = ymax - ymin;
    A_area =  bbox_size<DataType>(A);
    B_area =  bbox_size<DataType>(B);

    return w*h/(A_area + B_area - w*h);
}

template<typename T_BOX,typename T_SCORE,int DIM, int TSIZE>
__global__  void nmskernel(
        float nmsThres,
        float predThre,
        T_BOX* boxes_data,
        T_SCORE* scores_data,
        int* index,

        bool* afterNMS
)
{
    __shared__ bool kept[DIM*TSIZE];
    ONNXBbox<T_BOX> localBoxes[TSIZE];
    ONNXBbox<T_BOX> globalBox;
    T_SCORE  localSores[TSIZE];
    int      localIndex[TSIZE];
    T_SCORE  globalScore;
    int      globalIndex;
    // load
    const int current_idx = threadIdx.x; // + threadIdx.z*blockDim.y*blockDim.x + threadIdx.y*blockDim.x;
#pragma unroll
    for(int i=0; i<TSIZE; i++)
    {
        const int ref_idx = current_idx*TSIZE + i;
        localBoxes[i].xmin = (boxes_data)[ref_idx*4 + 0];
        localBoxes[i].ymin = (boxes_data)[ref_idx*4 + 1];
        localBoxes[i].xmax = (boxes_data)[ref_idx*4 + 2];
        localBoxes[i].ymax = (boxes_data)[ref_idx*4 + 3];
        localSores[i] = (scores_data)[ref_idx];
        localIndex[i]  = index[ref_idx];
        kept[current_idx*TSIZE + i] = (localSores[i] > predThre) && (localIndex[i] != 0);
    }

    __syncthreads();

    // compute
    for(int g=0; g<DIM*TSIZE; g++)
    {

        if(!kept[g])continue;
        globalBox.xmin = (boxes_data)[g*4 + 0];
        globalBox.ymin = (boxes_data)[g*4 + 1];
        globalBox.xmax = (boxes_data)[g*4 + 2];
        globalBox.ymax = (boxes_data)[g*4 + 3];
        globalIndex = index[g];
        globalScore = ((T_SCORE*)scores_data)[g];
        for(int i=0;i<TSIZE;i++)
        {
            if(kept[current_idx* TSIZE + i] && globalIndex == localIndex[i] && IOU(globalBox, localBoxes[i] )> nmsThres)
            {
                if(globalScore > localSores[i])
                {
                    kept[current_idx*TSIZE + i] = false;
                }
            }
        }
    }
    __syncthreads();
    // store
#pragma unroll
    for(int i=0;i<TSIZE;i++)
    {
        afterNMS[current_idx*TSIZE + i] = kept[current_idx*TSIZE+i];
    }
    __syncthreads();

}


template<typename T_BOXES, typename T_SCORES, int num>
void nmsLaunch(
        hipStream_t stream,
        const float predThre,
        const float nmsThre,
        void* bboxes,
        void* scores,
        int* indexs,
        bool* kept)
{


#define NMS(tsize) nmskernel<T_BOXES, T_SCORES,num,(tsize)>

    void (*kernel[1])(float,float, T_BOXES*, T_SCORES*, int*, bool*) = {NMS(23)};

    kernel[0]<<<1,num,0,stream>>>(nmsThre, predThre, (T_BOXES*)bboxes, (T_SCORES*)scores, indexs, kept);
}

typedef void (*nmsFun)(hipStream_t, float, float, void*, void*, int*, bool*);



struct nmsLaunchConfig
{
    nvinfer1::DataType T_boxes;
    nvinfer1::DataType T_scores;
    nmsFun function;

    nmsLaunchConfig(
            nvinfer1::DataType T_box,
            nvinfer1::DataType T_score,
            nmsFun fun
    )
    {
        this->T_boxes = T_box;
        this->T_scores = T_score;
        this->function = fun;
    };
    bool operator==(const nmsLaunchConfig& other)
    {
        return false;
    }
};

#define FLOAT32 nvinfer1::DataType::kFLOAT

static std::array<nmsLaunchConfig, 1> nmsLCOptions =
        {
                nmsLaunchConfig(FLOAT32,FLOAT32,nmsLaunch<float,float,178> )
        };

void nms(
        hipStream_t stream,
        const float predThre,
        const float nmsThre,
        void* box_data,
        void* score_data,
        int* index_data,
        bool* kept
)
{
    return nmsLCOptions[0].function(
            stream,
            predThre,
            nmsThre,
            box_data,
            score_data,
            index_data,
            kept
    );
}